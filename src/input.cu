#include <iostream>
#include <map>
#include <algorithm>
#include <cassert>
#include <thrust/host_vector.h>
#include <thrust/device_vector.h>
#include <thrust/sort.h>
#include <thrust/adjacent_difference.h>
#include <thrust/binary_search.h>

#include "constants.cuh"
#include "input.cuh"

using namespace std;

Graph getGraph()
{
    ios_base::sync_with_stdio(false);
    cin.tie(NULL);
    map<uint32_t, int> idxs;
    uint32_t a, b;
    thrust::host_vector<int> v1s_host; // starting node i.e. v1 -> v2
    thrust::host_vector<int> v2s_host; // end node
    thrust::host_vector<int> temp_edge(2);
    int idx = 0;
    int clique_size;
    cin >> clique_size;
    while (cin >> a)
    {
        cin >> b;
        auto temp = idxs.try_emplace(a, idx);
        temp_edge[0] = temp.first->second; // idx if new, a's already assigned idx otherwise
        if (temp.second)
        { // if insertion took place increment
            idx++;
        }
        // do same for b
        temp = idxs.try_emplace(b, idx);
        temp_edge[1] = temp.first->second;
        if (temp.second)
        {
            idx++;
        }
        v1s_host.insert(v1s_host.end(), temp_edge.cbegin(), temp_edge.cend());
        v2s_host.insert(v2s_host.end(), temp_edge.crbegin(), temp_edge.crend());
    }
    g_const::num_edges_host = v1s_host.size();
    g_const::num_vertices_host = idx;
    hipMemcpyToSymbol(HIP_SYMBOL(&g_const::num_edges_dev), &g_const::num_edges_host, sizeof g_const::num_edges_host);
    hipMemcpyToSymbol(HIP_SYMBOL(&g_const::num_vertices_dev), &g_const::num_vertices_host, sizeof g_const::num_vertices_host);
    thrust::device_vector<int> v1s_dev(v1s_host);
    thrust::device_vector<int> v2s_dev(v2s_host);
    auto edge_it = thrust::make_zip_iterator(thrust::make_tuple(v1s_dev.begin(), v2s_dev.begin()));
    thrust::sort(edge_it, edge_it + g_const::num_edges_host);

    // adding row pointer array using dense cummulative histogram,
    // based on https://github.com/NVIDIA/thrust/blob/master/examples/histogram.cu
    thrust::device_vector<int> row_pointer(g_const::num_vertices_host, 0);
    thrust::counting_iterator<int> idx_counter(0);
    thrust::upper_bound(v1s_dev.begin(), v1s_dev.end(),
                        idx_counter, idx_counter + idx,
                        row_pointer.begin() + 1);
    PRINTER(row_pointer);
    Graph res;
    res.v1s = v1s_dev;
    res.v2s = v2s_dev;
    res.row_ptr = row_pointer;
    return res;
}