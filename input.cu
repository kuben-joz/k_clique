#include <iostream>
#include <fstream>
#include <map>
#include <algorithm>
#include <cassert>
#include <thrust/host_vector.h>
#include <thrust/device_vector.h>
#include <thrust/sort.h>
#include <thrust/adjacent_difference.h>
#include <thrust/binary_search.h>

#include "constants.cuh"
#include "input.cuh"

using namespace std;

Graph getGraph(char *filename)
{
    ifstream in_file(filename);
    map<uint32_t, int> idxs;
    uint32_t a, b;
    thrust::host_vector<int> v1s_host; // starting node i.e. v1 -> v2
    thrust::host_vector<int> v2s_host; // end node
    thrust::host_vector<int> temp_edge(2);
    int idx = 0;
    while (in_file >> a)
    {
        in_file >> b;
        auto temp = idxs.try_emplace(a, idx);
        temp_edge[0] = temp.first->second; // idx if new, a's already assigned idx otherwise
        if (temp.second)
        { // if insertion took place increment
            idx++;
        }
        // do same for b
        temp = idxs.try_emplace(b, idx);
        temp_edge[1] = temp.first->second;
        if (temp.second)
        {
            idx++;
        }
        v1s_host.insert(v1s_host.end(), temp_edge.cbegin(), temp_edge.cend());
        v2s_host.insert(v2s_host.end(), temp_edge.crbegin(), temp_edge.crend());
    }
    g_const::num_edges_host = v1s_host.size();
    g_const::num_vertices_host = idx;
    HANDLE_ERROR(hipMemcpyToSymbol(HIP_SYMBOL(g_const::num_edges_dev), &g_const::num_edges_host, sizeof g_const::num_edges_host));
    HANDLE_ERROR(hipMemcpyToSymbol(HIP_SYMBOL(g_const::num_vertices_dev), &g_const::num_vertices_host, sizeof g_const::num_vertices_host));
    thrust::device_vector<int> v1s_dev(v1s_host);
    thrust::device_vector<int> v2s_dev(v2s_host);
    auto edge_it = thrust::make_zip_iterator(thrust::make_tuple(v1s_dev.begin(), v2s_dev.begin()));
    thrust::sort(edge_it, edge_it + g_const::num_edges_host);

    // adding row pointer array using dense cummulative histogram,
    // based on https://github.com/NVIDIA/thrust/blob/master/examples/histogram.cu
    thrust::device_vector<int> row_pointer(g_const::num_vertices_host + 1, 0);
    thrust::counting_iterator<int> idx_counter(0);
    thrust::upper_bound(v1s_dev.begin(), v1s_dev.end(),
                        idx_counter, idx_counter + idx,
                        row_pointer.begin() + 1);
    Graph res;
    res.v1s = v1s_dev;
    res.v2s = v2s_dev;
    res.row_ptr = row_pointer;
    return res;
}